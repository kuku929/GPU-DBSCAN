#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#define WHITE_VAL 255
#define SIZE 64


struct Point{
    Point() = default;
    Point(int i, int j): x(i), y(j){}
    int x, y;

    __device__ __host__ float euclidean_distance(Point &other){
            return sqrt((float)((this->x - other.x)*(this->x - other.x) + (this->y - other.y)*(this->y - other.y)));
    }
    __device__ __host__ int manhattan_distance(Point &other){
            return abs(this->x - other.x) + abs(this->y - other.y);
    }


};

__global__ void num_neighbors(int *count_list, Point *points, int no_of_nodes, float eps){
    int index = blockIdx.x*SIZE + threadIdx.x;
    if(index < no_of_nodes){
        int temp=0;
        for(int i=0;i < no_of_nodes; ++i){
            if(i == index)
                continue;
            if(points[index].euclidean_distance(points[i]) <= eps){
                temp++;
            }
        }
        count_list[index]=temp;
    }
}

__global__ void make_graph(int *adj_list, int *offset, Point *points, int no_of_nodes, int eps){
    int index = blockIdx.x*SIZE + threadIdx.x;

    // //debug
    // if(index == 0){
    //     printf("testing : %d\n", offset[1]);//points[0].euclidean_distance(points[1]));
    // }

    if(index < no_of_nodes){
        int curr_ind = 0;
        for(int i=0;i < no_of_nodes; ++i){
            if(i == index)
                continue;
            if(points[index].euclidean_distance(points[i]) <= eps){
                adj_list[offset[index] + curr_ind] = i;
                curr_ind++;
            }
        }
    }
}

class Graph{
    public:
        Graph() = default;
        ~Graph(){
            hipFree(adj_list);
            hipFree(dev_prefix);
            delete []prefix_sum;
        }
        Graph(cv::Mat &binary_image, float eps) : eps(eps){
            //filter black points
            find_nodes(binary_image);
            std::vector<int> neighbor_list = std::vector<int>(nodes.size(), 0);
            int *dev_neighbor_list;

            //allocate nodes on device
            Point *dev_nodes;
            hipMalloc(&dev_nodes, sizeof(Point)*nodes.size());
            hipMemcpy(dev_nodes, nodes.data(), sizeof(Point)*nodes.size(), hipMemcpyHostToDevice);
            // std::cout << (int)nodes[0].x << '\n';
            // std::cout <<"size : " <<  nodes.size() << '\n';
            hipMalloc(&dev_neighbor_list, sizeof(int)*neighbor_list.size());

            //find neighbors
            dim3 dim_block(SIZE, 1);
            dim3 dim_grid((nodes.size() + SIZE-1)/SIZE, 1);
            num_neighbors<<<dim_grid, dim_block>>>(dev_neighbor_list, dev_nodes, nodes.size(), eps);
            
            //back to host
            hipMemcpy(neighbor_list.data(), dev_neighbor_list, sizeof(int)*neighbor_list.size(), hipMemcpyDeviceToHost);

            // //debug
            // for(int i=0;i < nodes.size(); ++i){
            //     std::cout << neighbor_list[i] << ' ';
            // }
            // std::cout << '\n';

            //allocating memory to adjacency list
            prefix_sum = new int[nodes.size()+1];
            prefix_sum[0] = 0;
            for(int i=1;i < nodes.size()+1; ++i){
                prefix_sum[i] = prefix_sum[i-1] + neighbor_list[i-1];
            }
            // std::cout << "prefix : " << prefix_sum[nodes.size()] << '\n';
            hipMalloc(&adj_list, sizeof(int)*(prefix_sum[nodes.size()]));
            hipMalloc(&dev_prefix, sizeof(int)*(nodes.size()+1)); 
            hipMemcpy(dev_prefix, prefix_sum, sizeof(int)*(nodes.size()+1), hipMemcpyHostToDevice);
            make_graph<<<dim_grid, dim_block>>>(adj_list, dev_prefix, dev_nodes, nodes.size(), eps);

            // //debug
            // std::cout << "adj list :\n";
            // int adj[prefix_sum[nodes.size()]];
            // hipMemcpy(adj, adj_list, sizeof(int)*prefix_sum[nodes.size()], hipMemcpyDeviceToHost);
            // for(int i=0;i < nodes.size(); ++i){
            //     for(int j=prefix_sum[i]; j < prefix_sum[i+1]; ++j){
            //         std::cout << adj[j] << ' ';
            //     }
            //     std::cout << '\n';
            // }

            hipFree(dev_nodes);
            hipFree(dev_neighbor_list);

        }

        size_t size(){
            return nodes.size();
        }

        Point node(int index){
            return nodes[index];
        }
    private:
        std::vector<Point> nodes;
        int *adj_list; 
        int *dev_prefix;
        int *prefix_sum;
        float eps;

        void find_nodes(cv::Mat &img){
            uchar *row;
            cv::MatIterator_<uchar> itr, end;
                for(int i=0; i < img.rows; ++i){
                    row = img.ptr<uchar>(i);
                    for(int j=0; j < img.cols; ++j){
                        if(row[j] == WHITE_VAL){
                            nodes.push_back(Point(i,j));
                        }
                    }

                }
        }
        friend class DBSCAN;
};

__global__ void search(int *adj_list, int *offset, uchar *frontier, uchar *v, float eps, int min_pts, int no_of_nodes, int *true_count){
    int index = blockIdx.x*SIZE + threadIdx.x;
    if(index < no_of_nodes){
        if(frontier[index]){  //if node is a frontier
            frontier[index] = 0; 
            // v[index]=1; 
            for(int neighbor=offset[index];neighbor < offset[index+1]; ++neighbor){ //set all its neighbors as frontiers
                if(!v[adj_list[neighbor]]){
                    //if border point
                    if(offset[adj_list[neighbor]+1]-offset[adj_list[neighbor]] >= min_pts){
                        frontier[adj_list[neighbor]] = 1;
                    }
                    v[adj_list[neighbor]] = 1;
                }
            }
        }

        //the first thread sums the frontier array
        int sum=0;
        if(index == 0){
            for(int i=0;i < no_of_nodes; ++i){
                sum+=frontier[i];
            }
            *true_count = sum;
        }
    }
}


__global__ void reset(int node, uchar *frontier, uchar *v){
    /*
    * kernel code to do bfs
    */
        int index = blockIdx.x*SIZE + threadIdx.x;

        int val=0;
        if(index == node)
            val=1;
        frontier[index] = val;
        v[index] = val;
}
class DBSCAN{
    public:
        DBSCAN(Graph *g, float eps, int min_pts) : graph(g), no_nodes(g->nodes.size()), eps(eps), min_pts(min_pts){
            //unified memory
            // hipMallocManaged(&visited, sizeof(int)*no_nodes);
            // hipMallocManaged(&labels, sizeof(int)*no_nodes);

            visited = new uchar[no_nodes];
            labels = new uchar[no_nodes];
            hipMallocManaged(&true_count, sizeof(int));
        }

        ~DBSCAN(){
            delete []visited;
            delete []labels;
            hipFree(true_count);
        }

        void identify_cluster(float eps, int min_pts){
            int cluster_id = 1;
            for(int i=0;i < no_nodes; ++i){
                visited[i] = 0;
                labels[i]=0;
            }        

            // allocating memory
            uchar *frontier; 
            uchar *v; 
            hipMalloc(&frontier, sizeof(uchar)*no_nodes);
            hipMalloc(&v, sizeof(uchar)*no_nodes);
            int neighbors;
            for(int node=0;node < no_nodes; ++node){
                neighbors = graph->prefix_sum[node+1] - graph->prefix_sum[node];
                if(!visited[node] && neighbors >= min_pts){
                    // std::cout << "n : " << neighbors << '\n';
                    // std::cout << "hi";
                    visited[node] = 1;
                    labels[node] = cluster_id; 
                    bfs(frontier, v, node, eps, min_pts, cluster_id++);
                }
            }
            hipFree(frontier);
            hipFree(v);

        }
        void show_labels(){
            std::cout << "labels :\n";
            for(int i=0;i < no_nodes; ++i){
                std::cout << (int)labels[i] << ' ';
            }
            std::cout << '\n';
        }

        uchar label(int index){
            return labels[index];
        }
    private:
        Graph *graph;
        int no_nodes;
        uchar *visited;
        uchar *labels;
        float eps;
        int min_pts;
        int *true_count;

        void bfs(uchar *frontier, uchar *v, int node, float eps, int min_pts, int cluster_id){
            /*
            * start from a node and do bfs
            */

            // //debug
            // int adj[graph->prefix_sum[graph->nodes.size()]];
            // hipMemcpy(adj, graph->adj_list, sizeof(int)*graph->prefix_sum[graph->nodes.size()], hipMemcpyDeviceToHost);
            // for(int i=0;i < graph->nodes.size(); ++i){
            //     for(int j=graph->prefix_sum[i]; j < graph->prefix_sum[i+1]; ++j){
            //         std::cout << adj[j] << ' ';
            //     }
            //     std::cout << '\n';
            // }


            dim3 dim_block(SIZE, 1);
            dim3 dim_grid((no_nodes + SIZE-1)/SIZE);
            reset<<<dim_grid,dim_block>>>(node, frontier, v);
            *true_count = 1;

            //debug
            // int counter=1;

            while(*true_count){
                search<<<dim_grid, dim_block>>>(graph->adj_list, graph->dev_prefix, frontier, v, eps, min_pts, no_nodes, true_count);
                hipDeviceSynchronize();
                // std::cout << *true_count << '\n';

                //debug
                // if(counter == 10){
                //     break;
                // }
                // counter++;
                
            }
            //back to host
            uchar V[no_nodes];
            hipMemcpy(V, v, sizeof(uchar)*no_nodes, hipMemcpyDeviceToHost);        

            for(int node=0;node < no_nodes; ++node){
                if(V[node]){
                    labels[node] = cluster_id;
                    visited[node] = 1;
                }
            }
        }
};

int main(){
    cv::Mat img = imread("/home/krutarth/Desktop/gdbscan/test.jpeg", cv::IMREAD_COLOR);
    cv::Mat grey_img;
    cv::Mat binary_img;
    cvtColor(img, grey_img, cv::COLOR_BGR2GRAY);
    threshold(grey_img, binary_img, 200, 255, cv::THRESH_BINARY);
    float eps = 10.0f;
    int min_pts = 10;

    Graph graph(binary_img, eps);
    DBSCAN scanner(&graph, 0.1f, 10);
    scanner.identify_cluster(eps, min_pts);

    scanner.show_labels();

    cv::Mat final(binary_img.size(), CV_8UC3, cv::Scalar(0));
    std::cout << final.rows << ' '<< final.cols << '\n';
    std::cout << binary_img.rows << ' '<< binary_img.cols << '\n';
    int max_label_ = 0;
    for(int i=0;i < graph.size(); ++i){
         if(scanner.label(i) > max_label_){
            max_label_ = scanner.label(i);
         }
    }

    // int count[max_label_+1]={0};
    // for(int i=0;i < graph.size(); ++i){
    //    count[scanner.label(i)]++; 
    // }
    // int max_count = 0;
    // int max_count_label= 0;
    // for(int i=1;i <= max_label_; ++i){
    //     if(count[i] > max_count)
    //         max_count = count[i];
    //         max_count_label= i;
    // }
    // std::cout << max_count << '\n';
    // for(int i=0;i < graph.size(); ++i){
    //     if(scanner.label(i) == max_count_label)
    //         final.at<uchar>(graph.node(i).x, graph.node(i).y) = 255;
    // }

    uchar color = 255;
    uchar color_diff = (255)/max_label_;
    for(int label=1; label <= max_label_; ++label){
        for(int j=0; j < graph.size(); ++j){
            if(scanner.label(j) == label){
		    final.at<uchar>(graph.node(j).x, graph.node(j).y*3) = color;
		    final.at<uchar>(graph.node(j).x, graph.node(j).y*3+1) = color/2;
		    final.at<uchar>(graph.node(j).x, graph.node(j).y*3+2) = color/3;
            }
        }
        color-=color_diff;
    }

    // std::cout << "debug : \n";
    // for(int j=0; j < graph.size(); ++j){
    //     final.at<uchar>(graph.node(j).x, graph.node(j).y) = 255;
    //     std::cout << graph.node(j).x << ' '<< graph.node(j).y << ' ';
    // }

    cv::namedWindow("final", cv::WINDOW_FULLSCREEN);
    // cv::namedWindow("binary", cv::WINDOW_NORMAL);
    imshow("final", final);
    // imshow("binary", binary_img);
    cv::waitKey(0);
    // return 0;

}
